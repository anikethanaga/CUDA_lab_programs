
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>


#define T 1024 // max threads per block


__global__ void vecAdd(int *a,int *b,int *c,int N);

int main(){
  int N=2048;
  int curr=2;
  printf("-----------------------------------------\n");
  while(N<=T*13){
  int a[N], b[N], gpu_add[N],cpu_add[N];
  int *dev_a,*dev_b,*dev_c;
  float time_gpu,time_cpu,timeindex,timeinit;

  for(int i=0;i<N;i++){
    a[i]=i+i;
    b[i]=i*i;
  }

  int size=N*sizeof(int);

  hipMalloc((void**) &dev_a,size);
  hipMalloc((void**) &dev_b,size);
  hipMalloc((void**) &dev_c,size);


  hipEvent_t startinit,endinit;
  hipEventCreate(&startinit);
  hipEventCreate(&endinit);
  hipEventRecord(startinit, 0);

  hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);

  hipEventRecord(endinit, 0);
  hipEventSynchronize(endinit);
  hipEventElapsedTime(&timeinit, startinit, endinit);


  hipEvent_t gpu_start,gpu_end;
  hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_end);
	hipEventRecord(gpu_start, 0);


  vecAdd<<<(int)(N+T)/T,T>>>(dev_a,dev_b,dev_c,N);

  hipDeviceSynchronize();
	hipEventRecord(gpu_end, 0);
	hipEventSynchronize(gpu_end);
	hipEventElapsedTime(&time_gpu, gpu_start, gpu_end);


  hipEvent_t startindex,endindex;
  hipEventCreate(&startindex);
	hipEventCreate(&endindex);
	hipEventRecord(startindex, 0);

  hipMemcpy(gpu_add,dev_c,size,hipMemcpyDeviceToHost);

  hipEventRecord(endindex, 0);
	hipEventSynchronize(endindex);
	hipEventElapsedTime(&timeindex, startindex, endindex);

  clock_t cpu_start,cpu_end;
  cpu_start=clock();
  for(int i=0;i<N;i++){
    cpu_add[i]=a[i]+b[i];
  }
  cpu_end=clock();

  timeinit/=1000;
  timeindex/=1000;
  time_gpu/=1000;
  time_cpu=float(cpu_end-cpu_start)/float(CLOCKS_PER_SEC);

  printf("Time for sending initial data from host to device : %f\t sec\n",timeinit);
  printf("Cuda program launched with %d block and %d threads\n",(int)(N+T)/T,T);
  printf("Time for sending calculated data from device to host : %f\t sec\n",timeindex);
  printf("GPU Time:%f seconds\n",time_gpu);
  printf("CPU Time:%f seconds\n",time_cpu);

  int flag=1;
  for(int i=0;i<N;i++){
    //aprintf("%d - %d - %d\n",gpu_add[i],cpu_add[i],i);
    if(gpu_add[i]!=cpu_add[i]){
      flag=0;
      break;
    }
  }

  if(flag){
    printf("TEST PASSED\n");
    printf("SPEED UP:%f\n",time_cpu/time_gpu);
  }
  else{
    printf("TEST FAILED\n");
  }


  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  printf("---------------------------------------------------------\n");
  curr++;
  N=T*curr;
  }
  exit(0);

}

__global__ void vecAdd(int *a,int *b,int *c,int N){
  int i=blockIdx.x*blockDim.x+threadIdx.x;

  if(i<N){
    c[i]=a[i]+b[i];
  }
}

