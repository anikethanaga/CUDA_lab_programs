#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>


#define BLOCK_DIM 16


__global__ void matrixMult(int *a,int *b,int *c,int width);


int main(){
  int curr=2;
int N=BLOCK_DIM*curr;
  printf("------------------------------------------\n");
  while(N<=BLOCK_DIM*16){
  int a[N][N], b[N][N], gpu_mul[N][N],cpu_mul[N][N];
  int *dev_a, *dev_b, *dev_c;
  float time_gpu,time_cpu,timeindex,timeinit;

  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      a[i][j]=i+j;
      b[i][j]=i*j;
    }
  }

  int size=N*N*sizeof(int);
  hipMalloc((void**) &dev_a,size);
  hipMalloc((void**) &dev_b,size);
  hipMalloc((void**) &dev_c,size);

  hipEvent_t startinit,endinit;
  hipEventCreate(&startinit);
  hipEventCreate(&endinit);
  hipEventRecord(startinit, 0);

  hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);

  hipEventRecord(endinit, 0);
  hipEventSynchronize(endinit);
  hipEventElapsedTime(&timeinit, startinit, endinit);



  hipEvent_t gpu_start,gpu_end;
  hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_end);
	hipEventRecord(gpu_start, 0);

  dim3 dimBlock(BLOCK_DIM,BLOCK_DIM);
  dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));

  matrixMult<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c,N);

  hipDeviceSynchronize();
	hipEventRecord(gpu_end, 0);
	hipEventSynchronize(gpu_end);
	hipEventElapsedTime(&time_gpu, gpu_start, gpu_end);


  hipEvent_t startindex,endindex;
  hipEventCreate(&startindex);
	hipEventCreate(&endindex);
	hipEventRecord(startindex, 0);

  hipMemcpy(gpu_mul,dev_c,size,hipMemcpyDeviceToHost);

  hipEventRecord(endindex, 0);
	hipEventSynchronize(endindex);
	hipEventElapsedTime(&timeindex, startindex, endindex);

  clock_t cpu_start,cpu_end;
  cpu_start=clock();

  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      int sum=0;
      for(int k=0;k<N;k++){
        sum+=a[i][k]*b[k][j];
      }
      cpu_mul[i][j]=sum;
    }
  }

  cpu_end=clock();

  timeinit/=1000;
  timeindex/=1000;
  time_gpu/=1000;
  time_cpu=float(cpu_end-cpu_start)/float(CLOCKS_PER_SEC);

  printf("Time for sending initial data from host to device : %f\t sec\n",timeinit);
  printf("Cuda program launched with %d blocks and %d threads\n",(int)ceil(N/dimBlock.x)*(int)ceil(N/dimBlock.y),BLOCK_DIM*BLOCK_DIM);
  printf("Time for sending calculated data from device to host : %f\t sec\n",timeindex);
  printf("GPU Time:%f seconds\n",time_gpu);
  printf("CPU Time:%f seconds\n",time_cpu);

  int flag=1;
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      if(gpu_mul[i][j]!=cpu_mul[i][j]){
        flag=0;
        break;
      }
    }
  }

  if(flag){
    printf("TEST PASSED\n");
    printf("SPEED UP:%f\n",time_cpu/time_gpu);
  }
  else{
    printf("TEST FAILED\n");
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  printf("------------------------------------------\n");
  curr++;
  N=BLOCK_DIM*curr;
  }
}

__global__ void matrixMult(int *a,int *b,int *c,int width){
  int k,sum=0;

  int col=threadIdx.x+blockDim.x*blockIdx.x;
  int row=threadIdx.y+blockDim.y*blockIdx.y;

  if(col<width && row<width){
    for(k=0;k<width;k++)
      sum+=a[row*width+k]*b[k*width+col];
    c[row*width+col]=sum;
  }
}

