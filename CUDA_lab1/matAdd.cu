#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>


#define BLOCK_DIM 16


__global__ void matrixAdd(int *a,int *b,int *c,int N);

int main(){
  int curr=2;
  int N=BLOCK_DIM*curr;
  printf("------------------------------------------\n");
  while(N<=BLOCK_DIM*16){
  int a[N][N], b[N][N], gpu_sum[N][N],cpu_sum[N][N];
  int *dev_a, *dev_b, *dev_c;
  float time_gpu,time_cpu,timeindex,timeinit;

  int size=N*N*sizeof(int);
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      a[i][j]=i+j;
      b[i][j]=i-j;
    }
  }


  hipMalloc((void**) &dev_a,size);
  hipMalloc((void**) &dev_b,size);
  hipMalloc((void**) &dev_c,size);

  hipEvent_t startinit,endinit;
  hipEventCreate(&startinit);
  hipEventCreate(&endinit);
  hipEventRecord(startinit, 0);

  hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);

  hipEventRecord(endinit, 0);
  hipEventSynchronize(endinit);
  hipEventElapsedTime(&timeinit, startinit, endinit);



  hipEvent_t gpu_start,gpu_end;
  hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_end);
	hipEventRecord(gpu_start, 0);

  dim3 dimBlock(BLOCK_DIM,BLOCK_DIM);
  dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));
  matrixAdd<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c,N);

  hipDeviceSynchronize();
	hipEventRecord(gpu_end, 0);
	hipEventSynchronize(gpu_end);
	hipEventElapsedTime(&time_gpu, gpu_start, gpu_end);

  hipEvent_t startindex,endindex;
  hipEventCreate(&startindex);
	hipEventCreate(&endindex);
	hipEventRecord(startindex, 0);

  hipMemcpy(gpu_sum,dev_c,size,hipMemcpyDeviceToHost);

  hipEventRecord(endindex, 0);
	hipEventSynchronize(endindex);
	hipEventElapsedTime(&timeindex, startindex, endindex);

  clock_t cpu_start,cpu_end;
  cpu_start=clock();
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      cpu_sum[i][j]=a[i][j]+b[i][j];
    }
  }
  cpu_end=clock();

  timeinit/=1000;
  timeindex/=1000;
  time_gpu/=1000;
  time_cpu=float(cpu_end-cpu_start)/float(CLOCKS_PER_SEC);


  printf("Time for sending initial data from host to device : %f\t sec\n",timeinit);
  printf("Cuda program launched with %d blocks and %d threads\n",(int)ceil(N/dimBlock.x)*(int)ceil(N/dimBlock.y),BLOCK_DIM*BLOCK_DIM);
  printf("Time for sending calculated data from device to host : %f\t sec\n",timeindex);
  printf("GPU Time:%f seconds\n",time_gpu);
  printf("CPU Time:%f seconds\n",time_cpu);

  int flag=1;
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      if(gpu_sum[i][j]!=cpu_sum[i][j]){
        flag=0;
        break;
      }
    }
  }

  if(flag){

    printf("TEST PASSED\n");
    printf("SPEED UP:%f\n",time_cpu/time_gpu);
  }
  else{
    printf("TEST FAILED\n");
  }
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  printf("------------------------------------------\n");
  curr++;
  N=BLOCK_DIM*curr;
}

}

__global__ void matrixAdd(int *a,int *b,int *c,int N){
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  int row=blockIdx.y*blockDim.y+threadIdx.y;

  int index=col+row*N;
  if(col<N && row<N){
    c[index]=a[index]+b[index];
  }
}

